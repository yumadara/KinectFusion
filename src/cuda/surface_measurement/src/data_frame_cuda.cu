#include "hip/hip_runtime.h"
#include <data_frame_cuda.h>

#include <surface_measurement_utils_cuda.h>

#include <iostream>

namespace kinect_fusion {

FrameDataCuda::FrameDataCuda(const Eigen::Matrix3f& cameraIntrinstics, std::size_t height, std::size_t width)
    {
        for (Level level = 0U; level < NUMBER_OF_LEVELS; level++) {
            
            std::cout << "Level index: " << level << std::endl;

            std::size_t scale = std::pow(2U, level);

            std::size_t current_height = height / scale;
            std::size_t current_width = width / scale;

            m_cameraIntrinstics[level] = computeLevelCameraIntrinstics(cameraIntrinstics, level);

            m_filteredDepthMaps[level] = Map2DfCuda(current_height, current_width);
            m_surfaces[level] = SurfaceCuda(current_height, current_width);
            
            std::cout << "Surface height: " << m_surfaces[level].getHeight() << std::endl;
        } 
    }

    void FrameDataCuda::updateValues(Map2Df& depths) {
        m_rowDepthMap = Map2DfCuda(depths);
        
        applyBiliteralFilter(m_rowDepthMap, m_filteredDepthMaps[0]); // Do not filter now
        // m_filteredDepthMaps[0] = depths;
        for (std::size_t i = 0; i < NUMBER_OF_LEVELS - 1; i++) {
            subsample(m_filteredDepthMaps[i], m_filteredDepthMaps[i + 1]);
        }
        // for (std::size_t i = 0; i < NUMBER_OF_LEVELS; i++) {
        //     fillVertexMap(m_filteredDepthMaps[i], m_cameraIntrinstics[i], m_surfaces[i].getVertexMap());
        //     // fillNormalMap(m_surfaces[i].getVertexMap(), m_surfaces[i].getNormalMap());
        // }
    }

} // namespace kinect_fusion