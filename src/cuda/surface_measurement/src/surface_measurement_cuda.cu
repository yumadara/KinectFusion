#include "hip/hip_runtime.h"
#include <surface_measurement_utils_cuda.h>

#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda.hpp>

namespace kinect_fusion {

// __global__
// void fillVertexMapKernel(const Map2DfCuda& depths, Map2DVector3fCuda& vertexMap, float fovX, float fovY, float cX, float cY) {
//     // Get depth intrinsics.
//     const int row = blockIdx.x * blockDim.x + threadIdx.x;
//     const int col = blockIdx.y * blockDim.y + threadIdx.y;

//     std::size_t idx = depths.getIndex(row, col); // linearized index
//     float depth = depths.get(idx);
//     if (depth == MINF) {
//         vertexMap.set(idx, Vector3f(MINF, MINF, MINF));
//     }
//     else {
//         // Back-projection to camera space.
//         vertexMap.set(idx, Vector3f((col - cX) / fovX * depth, (row - cY) / fovY * depth, depth));
//     }
// }

// void fillVertexMap(const Map2DfCuda& depths, const Eigen::Matrix3f& depthIntrinsics, Map2DVector3fCuda& vertexMap)
// {
//     float fovX = depthIntrinsics(0, 0);
//     float fovY = depthIntrinsics(1, 1);
//     float cX = depthIntrinsics(0, 2);
//     float cY = depthIntrinsics(1, 2);

//     dim3 threads(32, 32);
//     dim3 blocks((depths.getNumberOfRows() + threads.x - 1) / threads.x, 
//                     (depths.getNumberOfColumns() + threads.y - 1) / threads.y);
//     fillVertexMapKernel<<<blocks, threads>>>(depths, vertexMap, fovX, fovY, cX, cY);
//     hipDeviceSynchronize();
// }

// __global__
// void fillNormalMapKernel(const Map2DVector3fCuda& vertexMap, Map2DVector3fCuda& normalMap) {

//     const int row = blockIdx.x * blockDim.x + threadIdx.x;
//     const int col = blockIdx.y * blockDim.y + threadIdx.y;

//     std::size_t idx = vertexMap.getIndex(row, col); // linearized index

//     auto firstVector{vertexMap.get(idx + vertexMap.getWidth()) - vertexMap.get(idx)};
//     auto secondVector{vertexMap.get(idx + 1) - vertexMap.get(idx)};

//     if (!firstVector.allFinite() || !secondVector.allFinite()) {
//         normalMap.get(idx) = Vector3f(MINF, MINF, MINF);
//     } else {
//         normalMap.get(idx) = firstVector.cross(secondVector);
//         normalMap.get(idx).normalize();
//     }       
// }

// void fillNormalMap(const Map2DVector3fCuda& vertexMap, Map2DVector3fCuda& normalMap) {
//     dim3 threads(32, 32);
//     dim3 blocks((vertexMap.getNumberOfRows() + threads.x - 1) / threads.x, 
//                     (vertexMap.getNumberOfColumns() + threads.y - 1) / threads.y);
//     fillNormalMapKernel<<<blocks, threads>>>(vertexMap, normalMap);
//     hipDeviceSynchronize();
// }

__global__
void subsampleKernel(const Map2DfCuda& previousDepthMap, Map2DfCuda& nextDepthMap)
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    const float centerPixel = previousDepthMap.get(row, col);
    float& newPixel{nextDepthMap.get(row / 2, col / 2)};

    if (centerPixel == MINF) {
        newPixel = MINF;
        return;
    }

    int topNeighbourRow = row - 1;
    int leftNeighbourCol = col - 1;
    int numberOfPixels = 9;

    if (col == 0) {
        leftNeighbourCol = 0;
        numberOfPixels = 6;
    }
    if (row == 0){
        topNeighbourRow = 0;
        numberOfPixels = 6;
    }
    if (col == 0 && row == 0)
    {
        numberOfPixels = 4;
    }

    newPixel = 0;
    for (int neighbourRow = topNeighbourRow; neighbourRow <= row + 1; neighbourRow++) {
        for (int neighbourCol = leftNeighbourCol; neighbourCol <= col + 1; neighbourCol++) {
            float neighbourValue = previousDepthMap.get(neighbourRow, neighbourCol);
            if (std::abs(neighbourValue - centerPixel) > MAX_DISTANCE) {
                numberOfPixels--;
            } else {
                newPixel += neighbourValue;
            }
        }                 
    }
    newPixel /= numberOfPixels;
}

void subsample(const Map2DfCuda& previousDepthMap, Map2DfCuda& nextDepthMap) {
    dim3 threads(32, 32);
    dim3 blocks((previousDepthMap.getNumberOfRows() + threads.x - 1) / threads.x, 
                    (previousDepthMap.getNumberOfColumns() + threads.y - 1) / threads.y);

    subsampleKernel<<<blocks, threads>>>(previousDepthMap, nextDepthMap);
    hipDeviceSynchronize();
}

void applyBiliteralFilter(Map2DfCuda& unfilteredMap, Map2DfCuda& filteredMap) {
    const float* a = thrust::raw_pointer_cast(unfilteredMap.data());
    const cv::cudev::GpuMat_<float> cvUnfilteredMap{unfilteredMap.getNumberOfColumns(), 
                    unfilteredMap.getNumberOfRows(), a};
    
    a = thrust::raw_pointer_cast(unfilteredMap.data());
    cv::cudev::GpuMat_<float> cvFilteredMap{filteredMap.getNumberOfColumns(), filteredMap.getNumberOfRows(), 
                a};

    constexpr float BIG_NEGATIVE_NUMBER = -10000.0; // -infinity leads to nan, so use

    // for (std::size_t i = 0; i < unfilteredMap.size(); i++) {
    //     if (unfilteredMap[i] == MINF) {
    //         unfilteredMap[i] = BIG_NEGATIVE_NUMBER;
    //     }
    // }
    cv::cuda::Stream stream;
    cv::bilateralFilter(cvUnfilteredMap, cvFilteredMap, FILTER_SIZE, SIGMA_DEPTH, SIGMA_SPACE, stream);
    // for (std::size_t i = 0; i < unfilteredMap.size(); i++) {
    //     if (unfilteredMap[i] == BIG_NEGATIVE_NUMBER) {
    //         unfilteredMap[i] = MINF;
    //         filteredMap[i] = MINF;
    //     }
    // }
}
} // namespace kinect_fusion